#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: SeqLabeler.cu
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Tue 14 Jun 2016 03:33:34 PM CST
 ************************************************************************/
#include <chrono>

#include "SeqLabeler.h"

SeqLabeler::SeqLabeler(bool b_train) {
    b_train_ = b_train;

    feature_extractor_ptr_.reset(new ChunkerFeatureExtractor());
    transition_system_ptr_.reset(new ChunkerTransitionSystem());
}

void SeqLabeler::train(DataSet &training_set, DataSet &dev_set) {
    trainInit(&training_set);

    const int num_in = feature_extractor_ptr_->getTotalInputSize();
    const int num_hidden = FLAGS_hidden_size;
    const int num_out = transition_system_ptr_->action_factory_ptr_->total_action_num;
    const int batch_size = std::min(FLAGS_batch_size, static_cast<int>(greedy_example_ptrs_.size()));

    std::clog << "# Begin to construct training model..." << std::endl;
    Stream<gpu> *stream = NewStream<gpu>();
    Model<gpu> model(num_in, num_hidden, num_out, feature_extractor_ptr_->feature_types_, stream);
    model.randomInitialize();
    // Model<gpu>::readWordPreTrain(FLAGS_embedding_file, feature_extractor_ptr_->getWordDict(), model.featEmbs[feature_extractor_ptr_->c_word_dict_index_]);
    Model<gpu> adagrad_squares(num_in, num_hidden, num_out, feature_extractor_ptr_->feature_types_, stream);  // for adagrad updating
    std::clog << "# End to construct training model!" << std::endl;

    double best_fscore = -1;
    for (int iter = 1; iter <= FLAGS_max_training_iteration_num; iter++) {

        // record the cost time
        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container,
        // get the shuffled training data for the mini-batch training of this iteration
        std::random_shuffle(greedy_example_ptrs_.begin(), greedy_example_ptrs_.end());
        std::vector<std::shared_ptr<Example>> multiThread_miniBatch_data(greedy_example_ptrs_.begin(),
                                                                         greedy_example_ptrs_.begin() + batch_size);

        // cumulated gradients for updating
//        Model<gpu> batch_cumulateddddd_grads(num_in, num_hidden, num_out, feature_extractor_ptr_->feature_types_, stream);
        Model<gpu> gradients(num_in, num_hidden, num_out, feature_extractor_ptr_->feature_types_, stream);

        // // create the neural net for prediction
        std::shared_ptr<FeedForwardNNet<gpu>> nnet;
        nnet.reset(new FeedForwardNNet<gpu>(batch_size, num_in, num_hidden, num_out, &model));

        // // feature vector lists for action sequence
        FeatureVectors feature_vectors(batch_size);

        // // batch input of
        TensorContainer<cpu, 2, real_t> input(Shape2(batch_size, num_in));

        std::vector<std::vector<int>> valid_action_vectors(batch_size, std::vector<int>(num_out,0));

        TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(batch_size, num_out));

        // /*
        //  * init the input and predict output
        //  */
        input = 0.0;
        batch_predict_output = 0.0;

        // // fill the feature vectors for batch training

        // // prepare batch training data!
        for (int inst = 0; inst < batch_size; inst++) {
            auto e = greedy_example_ptrs_[inst];

            feature_vectors[inst] = e->feature_vector;
            valid_action_vectors[inst] = e->predict_labels;
        }

        feature_extractor_ptr_->returnInput(feature_vectors, model.featEmbs, input);

        nnet->ChunkForward(input, batch_predict_output, FLAGS_be_dropout);

        int total_correct_predict_action_num = 0;
        double loss = 0;
        for (int insti = 0; insti < batch_size; insti++) {
            int opt_act = -1;
            int gold_act = -1;

            std::vector<int> &valid_acts = valid_action_vectors[insti];

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[insti][i] > batch_predict_output[insti][opt_act]) {
                        opt_act = i;
                    }

                    if (valid_acts[i] == 1) {
                        gold_act = i;
                    }
                }
            }

            if (opt_act == gold_act) {
                total_correct_predict_action_num += 1;
            }

            real_t max_score = batch_predict_output[insti][opt_act];
            real_t gold_score = batch_predict_output[insti][gold_act];

            const int act_num = valid_acts.size();
            std::vector<real_t> lx(act_num, 0.0);
            std::vector<real_t> x_bar(act_num, 0.0);
            std::vector<real_t> x_barExp(act_num, 0.0);
            std::vector<real_t> y_bar(act_num, 0.0);
            std::vector<real_t> z_bar(act_num, 0.0);
            real_t tloss = 0.0;
            real_t Q = 0.0;
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    x_bar[i] = batch_predict_output[insti][i] - max_score;
                    x_barExp[i] = exp(x_bar[i]);
                    Q += x_barExp[i];
                }
            }
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    y_bar[i] = x_barExp[i] / Q;
                }
            }
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    if (y_bar[i] <= 0.5) {
                        z_bar[i] = 1.0 - y_bar[i];
                    } else {
                        z_bar[i] = (Q - x_barExp[i]) / Q;
                    }
                }
            }
            for (int i = 0; i < act_num; i++) {
                real_t tbar = (i == gold_act) ? 1.0 : 0.0;

                if (valid_acts[i] >= 0) {
                    if (y_bar[i] <= 0.5) {
                        lx[i] = y_bar[i] - tbar;

                        tloss += tbar * std::log(y_bar[i]);
                    } else {
                        lx[i] = ( 1.0 - tbar ) - z_bar[i];

                        tloss += tbar * std::log( 1.0 - z_bar[i] );
                    }
                }
            }

            loss -= tloss;

            for (int i = 0; i < act_num; i++) {
                batch_predict_output[insti][i] = lx[i];
            }
        }

        batch_predict_output /= batch_size;

        nnet->ChunkBackprop(batch_predict_output);
        nnet->SubsideGradsTo(&gradients, feature_vectors);

        model.update(&g, &adagrad_squares);
        auto end = std::chrono::high_resolution_clock::now();

        if (iter % FLAGS_evaluate_per_iteration == 0) {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000000.0 ;
            std::clog << "[" << iter << "] totally train " << batch_size << " examples, time: " << time_used <<
            " average: " << batch_size / time_used << " examples/second!" << std::endl;
            double posClassificationRate = static_cast<double>(total_correct_predict_action_num) / batch_size;
            double regular_loss = 0.5 * FLAGS_regularization_rate * model.norm2();
            double avg_loss = (loss  + regular_loss) / batch_size;
            std::cerr << "current objective fun-score  : " << avg_loss << "\tclassfication rate: " << posClassificationRate << std::endl;
        }

        /*
         * do the evaluation in iteration of training
         * save the best resulting model
         */
        // if (iter % FLAGS_evaluate_per_iteration == 0) {
        //     // do the evaluation
        //     double dev_uas = test(dev_data, model, nnet.operator*());
        //     if (dev_uas > best_fscore){
        //         std::ofstream ofs(FLAGS_model_file);
        //         model.saveModel(ofs);
        //         ofs.close();

        //     }
        // }
    }
}

void SeqLabeler::greedyTrain(DataSet &training_set, DataSet &dev_set) {
}

double SeqLabeler::test(DataSet &test_data, Model<cpu> &model, FeedForwardNNet <gpu> &net) {

    return 0.0;
}

void SeqLabeler::trainInit(DataSet *training_set_ptr) {
    std::clog << "======================================"<<std::endl;
    std::clog << "Training Init!" << std::endl;
    std::clog << "Training Instance Num: " << training_set_ptr->getSize() << std::endl;
    std::clog << "======================================"<<std::endl;

    // prepare the handler for parsing
    std::clog << "## Begin to init the dictionaries..." << std::endl;
    feature_extractor_ptr_->getDictionaries(training_set_ptr);  // dictionary for feature index
    feature_extractor_ptr_->displayDict();

    std::clog << "## End to init the dictionaries!" << std::endl;

    std::clog << "## Begin to create feature types..." << std::endl;
    feature_extractor_ptr_->setFeatureTypes();
    feature_extractor_ptr_->displayFeatureTypes();
    std::clog << "## End to create feature types!" << std::endl;

    std::clog << "## Begin to init the transition system..." << std::endl;
    transition_system_ptr_->makeTransitions(feature_extractor_ptr_->getKnownLabelVector(), feature_extractor_ptr_->getKnownLabel2IndexMap());
    transition_system_ptr_->setHeadWordRule(std::shared_ptr<HeadWordRule>(new HeadWordRule(feature_extractor_ptr_->getTagDict(), feature_extractor_ptr_->getLabelDict())));
    std::clog << "## End to init the transition system!" << std::endl;

    std::clog << "## Begin to generate the training examples..." << std::endl;
    GreedyChunker greedy_chunker;
    greedy_chunker.generateTrainingExamples(transition_system_ptr_.get(),
                                            feature_extractor_ptr_.get(),
                                            static_cast<SeqLabelerDataSet*>(training_set_ptr),
                                            greedy_example_ptrs_);
    std::clog << "### Greedy training examples' size: " << greedy_example_ptrs_.size() << std::endl;
    std::clog << "## End to generate the training examples!" << std::endl;
}
