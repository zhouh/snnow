#include "hip/hip_runtime.h"
//
// Created by zhouh on 16-4-5.
//
#include <ctime>
#include <memory>

#include "DepParser.h"

using namespace mshadow;
using namespace mshadow::expr;

void printVector(std::vector<int> vec) {

    for (int i = 0; i < vec.size(); i++) {
        std::cout << "<" << i << ">=" << vec[i] << std::endl;
    }

}

/**
 * initialize the feature extractor and
 * transition system handlers for parser
 */
DepParser::DepParser(bool bTrain) {
    beam_size = FLAGS_beam_size;
    be_train = bTrain;
    trainsition_system_ptr.reset(new DepArcStandardSystem());
    feature_extractor_ptr.reset(new DepParseFeatureExtractor());
}

/**
 *  do the training init for formal training
 *
 *  0. init the feature type of this system
 *  1. get dictionary for the feature extractor
 *  2. init the transition system handler for the parser
 *  3. init the feature embedding handler
 */
void DepParser::trainInit(DataSet *training_data_ptr) {

    std::clog << "======================================" << std::endl;
    std::clog << "Training Init!" << std::endl;
    std::clog << "Training Instance Num: " << training_data_ptr->getSize() << std::endl;
    std::clog << "======================================" << std::endl;


    std::clog << "###Begin to init the feature types of this system: " << std::endl;


    // prepare the handler for parsing
    std::clog << "###Begin to init the dictionaries!" << std::endl;
    feature_extractor_ptr->getDictionaries(training_data_ptr);  // dictionary for feature index
    feature_extractor_ptr->displayDict();
    std::clog << "###End to init the dictionaries!" << std::endl;

    std::clog << "###Begin to create feature types!" << std::endl;

    /*
     * total 3 feature types for dependency parsing
     * word feature
     * tag feature
     * label feature
     */
    FeatureTypes feature_types;
    FeatureType word_feat_type(DepParseFeatureExtractor::word_string,
                               feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_word_dict_index],
                               feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_word_dict_index]->size(),
                               c_word_feature_dim);
    FeatureType tag_feat_type(DepParseFeatureExtractor::tag_string,
                              feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_tag_dict_index],
                              feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_tag_dict_index]->size(),
                              c_tag_feature_dim);
    FeatureType label_feat_type(DepParseFeatureExtractor::label_string,
                                feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_dep_label_dict_index],
                                feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_dep_label_dict_index]->size(),
                                c_label_feature_dim);
    feature_types.push_back(word_feat_type);
    feature_types.push_back(tag_feat_type);
    feature_types.push_back(label_feat_type);

    // set the feature types for feature handlers
    feature_extractor_ptr->setFeatureTypes(feature_types);
    FeatureVector::setFeatureTypes(feature_types);
    std::clog << "###End to create feature types!" << std::endl;


    // init transition system
    std::clog << "###Init the transition system!" << std::endl;
    trainsition_system_ptr->makeTransition(feature_extractor_ptr->getKnownDepLabelVector(),
                                           feature_extractor_ptr->getKnownDepLabelVectorMap());

    std::clog << "###Begin to generate the training examples!" << std::endl;

    feature_extractor_ptr->generateGreedyTrainingExamples(trainsition_system_ptr.get(),
                                                          static_cast<DepParseDataSet *>(training_data_ptr),
                                                          greedy_example_ptrs);
    std::clog << "Constructing dictionary and training examples done!" << std::endl;
}

void DepParser::train(DataSet *train_data_ptr, DataSet *dev_data_ptr) {

    // init training
    trainInit(train_data_ptr);

    /*
     * prepare for the neural networks, every parsing step maintains a specific net
     * because each parsing step has different updating gradients.
     */
    const int num_in = feature_extractor_ptr->getTotalInputSize();
    const int num_hidden = FLAGS_hidden_size;
    const int num_out = trainsition_system_ptr->action_factory_ptr->total_action_num;
//    const int beam_size = FLAGS_beam_size;
    const int batch_size = std::min(FLAGS_batch_size, static_cast<int>(greedy_example_ptrs.size()));
//    const bool be_dropout = FLAGS_dropout_prob;


    /*
     * create the model for training
     */
    std::clog << "###Begin to construct training model." << std::endl;
    Stream <gpu> *stream = NewStream<gpu>();
    Model<gpu> model(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, stream);
    model.randomInitialize();
    Model<gpu> adagrad_squares(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types,
                               stream);  // for adagrad updating
    std::clog << "###End to construct training model." << std::endl;


    double best_uas = -1;
    for (int iter = 1; iter <= FLAGS_max_training_iteration_num; iter++) {

        std::clog << "iteration\t" << iter << std::endl;

        // record the cost time
        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container,
        // get the shuffled training data for the mini-batch training of this iteration
        std::cout << "shuffle the training data." << std::endl;
        std::random_shuffle(greedy_example_ptrs.begin(), greedy_example_ptrs.end());
        int batch_example_index_end = std::min(batch_size, static_cast<int>(greedy_example_ptrs.size()));
        std::vector<std::shared_ptr<Example>> multiThread_miniBatch_data(greedy_example_ptrs.begin(),
                                                                         greedy_example_ptrs.begin() +
                                                                         batch_example_index_end);

        // cumulated gradients for updating
//        Model<gpu> batch_cumulated_grads(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, NULL);
        Model<gpu> gradients(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, stream);

        // create the neural net for prediction
        std::shared_ptr<FeedForwardNNet<gpu>> nnet;
        nnet.reset(new FeedForwardNNet<gpu>(batch_size, num_in, num_hidden, num_out, &model));

        // feature vector lists for action sequence
        FeatureVectors feature_vectors(multiThread_miniBatch_data.size());

        // batch input of
        TensorContainer<cpu, 2, real_t> input(Shape2(batch_size, num_in));

        std::vector<std::vector<int>> valid_action_vectors(multiThread_miniBatch_data.size());

        TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(batch_size, num_out));

        /*
         * init the input and predict output
         */
        input = 0.0;
        batch_predict_output = 0.0;

        // fill the feature vectors for batch training


        // prepare batch training data!
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {
            auto e = greedy_example_ptrs[inst];

            feature_vectors[inst] = e->feature_vector;
            valid_action_vectors[inst] = e->predict_labels;
        }

        feature_extractor_ptr->returnInput(feature_vectors, model.featEmbs, input);

//        for (int j = 0; j < input.size(0); ++j) {
//            for (int i = 0; i < input.size(1); ++i) {
//                std::cout << input[j][i] << "\t";
//            }
//            std::cout << std::endl;
//
//        }

        nnet->Forward(input, batch_predict_output, FLAGS_be_dropout);


        int total_correct_predict_action_num = 0;
        double loss = 0;
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {

            int opt_act = -1;
            int gold_act = -1;

            std::vector<int> &valid_acts = valid_action_vectors[inst];

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {

                    std::cout << "batch_predict_output\t"<<i<<"\t"<<batch_predict_output[inst][i] << std::endl;
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }

                    if (valid_acts[i] == 1) {
                        gold_act = i;
                    }
                }
            }

            std::cout << inst <<": opt\t"<<opt_act<< std::endl;
            std::cout << inst <<": gold\t"<<gold_act<< std::endl;

            if (opt_act == gold_act) {
                total_correct_predict_action_num += 1;
            }

            real_t max_score = batch_predict_output[inst][opt_act];
            real_t gold_score = batch_predict_output[inst][gold_act];

//            real_t sum = 0.0;
//
//            for (int i = 0; i < valid_acts.size(); i++) {
//                if (valid_acts[i] >= 0) {
//                    batch_predict_output[inst][i] = std::exp(batch_predict_output[inst][i] - max_score);
//                    sum += batch_predict_output[inst][i];
//                }
//            }
//
//            loss += (std::log(sum) - (gold_score - max_score)) / multiThread_miniBatch_data.size();
//
//            for (int i = 0; i < valid_acts.size(); i++) {
//                if (valid_acts[i] >= 0) {
//                    batch_predict_output[inst][i] = batch_predict_output[inst][i] / sum;
//                } else {
//                    batch_predict_output[inst][i] = 0.0;
//                }
//            }
//            batch_predict_output[inst][gold_act] -= 1.0;
//        }

            const int act_num = valid_acts.size();
            std::vector<real_t> lx(act_num, 0.0);
            std::vector<real_t> x_bar(act_num, 0.0);
            std::vector<real_t> x_barExp(act_num, 0.0);
            std::vector<real_t> y_bar(act_num, 0.0);
            std::vector<real_t> z_bar(act_num, 0.0);
            real_t tloss = 0.0;
            real_t Q = 0.0;
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    x_bar[i] = batch_predict_output[inst][i] - max_score;
                    x_barExp[i] = exp(x_bar[i]);
                    Q += x_barExp[i];
                }
            }
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    y_bar[i] = x_barExp[i] / Q;
                }
            }
            for (int i = 0; i < act_num; i++) {
                if (valid_acts[i] >= 0) {
                    if (y_bar[i] <= 0.5) {
                        z_bar[i] = 1.0 - y_bar[i];
                    } else {
                        z_bar[i] = (Q - x_barExp[i]) / Q;
                    }
                }
            }
            for (int i = 0; i < act_num; i++) {
                real_t tbar = (i == gold_act) ? 1.0 : 0.0;

                if (valid_acts[i] >= 0) {
                    if (y_bar[i] <= 0.5) {
                        lx[i] = y_bar[i] - tbar;

                        tloss += tbar * std::log(y_bar[i]);
                    } else {
                        lx[i] = (1.0 - tbar) - z_bar[i];

                        tloss += tbar * std::log(1.0 - z_bar[i]);
                    }
                }
            }

            loss -= tloss;

            for (int i = 0; i < act_num; i++) {
                batch_predict_output[inst][i] = lx[i];
            }
        }

//        batch_predict_output /= batch_size;

        batch_predict_output /= static_cast<real_t>(multiThread_miniBatch_data.size());

        nnet->Backprop(batch_predict_output);
        nnet->SubsideGradsTo(&gradients, feature_vectors); // add the gradients from the nets to the gradients
        model.update(&gradients,
                     &adagrad_squares); // update the gradient with adagrad, and update the parameters in the models
        auto end = std::chrono::high_resolution_clock::now();

        if (iter % FLAGS_evaluate_per_iteration == 0) {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000000.0;
            std::clog << "[" << iter << "] totally train " << batch_size << " examples, time: " << time_used <<
            " average: " << batch_size / time_used << " examples/second!" << std::endl;

            double posClassificationRate = static_cast<double>(total_correct_predict_action_num) / batch_size;
            double regular_loss = 0.5 * FLAGS_regularization_rate * model.norm2();
            double avg_loss = (loss + regular_loss) / batch_size;
            std::clog << "current objective fun-score  : " << avg_loss << "\tclassfication rate: " <<
            posClassificationRate <<

            /*
         * do the evaluation in iteration of training
         * save the best resulting model
         */
            std::cout << "###Test Begin###" << std::endl;
            // do the evaluation
            double dev_uas = test(dev_data_ptr, &model, nnet.get());
            std::clog << "Current Iteration UAS\t" << dev_uas << "%" << std::endl;
            if (dev_uas > best_uas) {
                std::ofstream ofs(FLAGS_model_file);
                model.saveModel(ofs);
                ofs.close();

            }
        }

    }


}

//===============================================================================
double DepParser::test(DataSet *test_data, Model<gpu> *model, FeedForwardNNet<gpu> *net) {

    const int num_in = feature_extractor_ptr->getTotalInputSize();
//    const int num_hidden = FLAGS_hidden_size;
    const int num_out = trainsition_system_ptr->action_factory_ptr->total_action_num;

    std::vector<DepParseTree> predict_trees(test_data->size);

    std::vector<DepParseTree> gold_dep_trees;

    std::shared_ptr<FeedForwardNNet<gpu>> nnet;
    nnet.reset(new FeedForwardNNet<gpu>(1, num_in, FLAGS_hidden_size, num_out, model));

    for (int inst = 0; inst < test_data->size; ++inst) {

        auto &tree_i = static_cast<DepParseTree & >(*(test_data->outputs[inst]));
        auto &input_i = static_cast<DepParseInput & >(*(test_data->inputs[inst]));

        feature_extractor_ptr->getCache(input_i);

        gold_dep_trees.push_back(tree_i);



        // n shift and n reduce, one more reduce action for root
        int total_act_num_one_sentence = (input_i.size() - 1) * 2;

        /*
         * cache the dependency label in the training set
         */
        std::vector<int> labelIndexCache(tree_i.size);
        int index = 0;
        for (auto iter = tree_i.nodes.begin(); iter != tree_i.nodes.end();
             iter++) {
            int labelIndex = feature_extractor_ptr->getLabelIndex(iter->label);

            if (labelIndex == -1) {
                std::cerr << "Dep label " << iter->label
                << " is not in labelMap!" << std::endl;
                exit(1);
            }

            labelIndexCache[index] = labelIndex;
            index++;
        }

        std::shared_ptr<DepParseState> state_ptr;
        state_ptr.reset(new DepParseState());

        state_ptr->len_ = input_i.size();
        state_ptr->initCache();
//        getCache(input_i);

//        state_ptr->toString();
        //for every state of a sentence
        for (int j = 0; !state_ptr->complete(); j++) {

            TensorContainer<cpu, 2, real_t> input(Shape2(1, num_in));
            TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(1, num_out));

            /*
             * init the input and predict output
             */
            input = 0.0;
            batch_predict_output = 0.0;

            std::vector<int> valid_acts;

            //get current state features
            FeatureVector fv = feature_extractor_ptr->getFeatureVectors(
                    static_cast<State *>(state_ptr.get()),
                    static_cast<Input *>(&input_i));
            FeatureVectors fvs;
            fvs.push_back(fv);

            //get current state valid actions
            trainsition_system_ptr->getValidActs(static_cast<State *>(state_ptr.get()), valid_acts);

//            printVector(valid_acts);

            feature_extractor_ptr->returnInput(fvs, model->featEmbs, input);

            nnet->Forward(input, batch_predict_output, FLAGS_dropout_prob);


            int opt_act = -1;
            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }
                }
            }

            trainsition_system_ptr->Move(static_cast<State *>(state_ptr.get()),
                                         DepParseShiftReduceActionFactory::action_table[opt_act].get());
//            std::cout<<"action:\t"<<static_cast<DepParseAction*>(DepParseShiftReduceActionFactory::action_table[opt_act].get())->toString(feature_extractor_ptr->dictionary_ptrs_table[feature_extractor_ptr->c_dep_label_dict_index])<<std::endl;
//            std::cout<<"label code\t"<<DepParseShiftReduceActionFactory::action_table[opt_act]->getActionLabel()<<std::endl;

//            state_ptr->toString();
        }


        // generate the predict tree from the complete state
        trainsition_system_ptr->GenerateOutput(static_cast<State *>(state_ptr.get()),
                                               static_cast<Input *>(&input_i),
                                               static_cast<Output *>( &(predict_trees[inst])));

        std::cout << "output tree:" << std::endl;
        std::cout << predict_trees[inst] << std::endl;


    }

    DepParseEvalb evalb;
    double result = evalb.evalb(predict_trees, gold_dep_trees);
    return result * 100;
}