#include "hip/hip_runtime.h"
//
// Created by zhouh on 16-4-5.
//
#include <ctime>
#include <memory>

#include "DepParser.h"

using namespace mshadow;
using namespace mshadow::expr;

/**
 * initialize the feature extractor and
 * transition system handlers for parser
 */
DepParser::DepParser(bool bTrain){
    beam_size = FLAGS_beam_size;
    be_train = bTrain;
    trainsition_system_ptr.reset(new DepArcStandardSystem());
    feature_extractor_ptr.reset(new DepParseFeatureExtractor());
}

/**
 *  do the training init for formal training
 *
 *  0. init the feature type of this system
 *  1. get dictionary for the feature extractor
 *  2. init the transition system handler for the parser
 *  3. init the feature embedding handler
 */
void DepParser::trainInit(DataSet &training_data) {

    std::clog << "======================================"<<std::endl;
    std::clog << "Training Init!" << std::endl;
    std::clog << "Training Instance Num: " << training_data.getSize() << std::endl;
    std::clog << "======================================"<<std::endl;


    std::clog << "###Begin to init the feature types of this system: " << std::endl;


    // prepare the handler for parsing
    std::clog << "###Begin to init the dictionaries!" << std::endl;
    feature_extractor_ptr->getDictionaries(training_data);  // dictionary for feature index
    feature_extractor_ptr->displayDict();
    std::clog << "###End to init the dictionaries!" << std::endl;

    std::clog << "###Begin to create feature types!" << std::endl;

    /*
     * total 3 feature types for dependency parsing
     * word feature
     * tag feature
     * label feature
     */
    FeatureTypes feature_types;
    FeatureType word_feat_type(DepParseFeatureExtractor::word_string,
                               feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_word_dict_index],
                               feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_word_dict_index]->size(),
                               c_word_feature_dim);
    FeatureType tag_feat_type(DepParseFeatureExtractor::tag_string,
                              feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_tag_dict_index],
                              feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_tag_dict_index]->size(),
                              c_tag_feature_dim);
    FeatureType label_feat_type(DepParseFeatureExtractor::label_string,
                                feature_extractor_ptr->feature_nums[DepParseFeatureExtractor::c_dep_label_dict_index],
                                feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_dep_label_dict_index]->size(),
                                c_label_feature_dim);
    feature_types.push_back(word_feat_type);
    feature_types.push_back(tag_feat_type);
    feature_types.push_back(label_feat_type);

    // set the feature types for feature handlers
    feature_extractor_ptr->setFeatureTypes(feature_types);
    FeatureVector::setFeatureTypes(feature_types);
    std::clog << "###End to create feature types!" << std::endl;


    // init transition system
    std::clog << "###Init the transition system!" << std::endl;
    trainsition_system_ptr->makeTransition(feature_extractor_ptr->getKnownDepLabelVector(),
                                           feature_extractor_ptr->getKnownDepLabelVectorMap());

    std::clog << "###Begin to generate the training examples!" << std::endl;

    feature_extractor_ptr->generateGreedyTrainingExamples(trainsition_system_ptr.get(), static_cast<DepParseDataSet&>(training_data), greedy_example_ptrs);
    std::clog << "Constructing dictionary and training examples done!" << std::endl;
}

void DepParser::train(DataSet &train_data, DataSet &dev_data) {

    // init training
    trainInit(train_data);

    /*
     * prepare for the neural networks, every parsing step maintains a specific net
     * because each parsing step has different updating gradients.
     */
    const int num_in = feature_extractor_ptr->getTotalInputSize();
    const int num_hidden = FLAGS_hidden_size;
    const int num_out = trainsition_system_ptr->action_factory_ptr->total_action_num;
//    const int beam_size = FLAGS_beam_size;
    const int batch_size = std::min(FLAGS_batch_size, static_cast<int>(greedy_example_ptrs.size()));
//    const bool be_dropout = FLAGS_dropout_prob;


    /*
     * create the model for training
     */
    std::clog << "###Begin to construct training model." << std::endl;
    Model<cpu> model(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, NULL);
    Model<cpu> adagrad_squares(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types,
                               NULL);  // for adagrad updating
    Stream <gpu> *stream = NewStream<gpu>();
    std::clog << "###End to construct training model." << std::endl;


    double best_uas = -1;
    for (int iter = 1; iter <= FLAGS_max_training_iteration_num; iter++) {

        // record the cost time
        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container,
        // get the shuffled training data for the mini-batch training of this iteration
        std::random_shuffle(greedy_example_ptrs.begin(), greedy_example_ptrs.end());
        int batch_example_index_end = std::min(batch_size, static_cast<int>(greedy_example_ptrs.size()));
        std::vector<std::shared_ptr<Example>> multiThread_miniBatch_data(greedy_example_ptrs.begin(),
        greedy_example_ptrs.begin() + batch_example_index_end);

        // cumulated gradients for updating
        Model<cpu> batch_cumulated_grads(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, NULL);
        Model<gpu> gradients(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, stream);

        // create the neural net for prediction
        std::shared_ptr<FeedForwardNNet<gpu>> nnet;
        nnet.reset(new FeedForwardNNet<gpu>(batch_size, num_in, num_hidden, num_out, &model));

        // feature vector lists for action sequence
        FeatureVectors feature_vectors(multiThread_miniBatch_data.size());

        // batch input of
        TensorContainer<cpu, 2, real_t> input(Shape2(batch_size, num_in));

        std::vector<std::vector<int>> valid_action_vectors(multiThread_miniBatch_data.size(), std::vector<int>(num_out,0));

        TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(batch_size, num_out));

        /*
         * init the input and predict output
         */
        input = 0.0;
        batch_predict_output = 0.0;

        // fill the feature vectors for batch training


        // prepare batch training data!
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {
            auto e = greedy_example_ptrs[inst];

            feature_vectors[inst] = e->feature_vector;
            valid_action_vectors[inst] = e->predict_labels;
        }

        feature_extractor_ptr->returnInput(feature_vectors, model.featEmbs, input);

        nnet->Forward(input, batch_predict_output, FLAGS_dropout_prob);


        int total_correct_predict_action_num = 0;
        double loss = 0;
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {

            int opt_act = -1;
            int gold_act = -1;

            std::vector<int> &valid_acts = valid_action_vectors[inst];

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }

                    if (valid_acts[i] == 1) {
                        gold_act = i;
                    }
                }
            }

            if (opt_act == gold_act) {
                total_correct_predict_action_num += 1;
            }

            real_t max_score = batch_predict_output[inst][opt_act];
            real_t gold_score = batch_predict_output[inst][gold_act];

            real_t sum = 0.0;

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    batch_predict_output[inst][i] = std::exp(batch_predict_output[inst][i] - max_score);
                    sum += batch_predict_output[inst][i];
                }
            }

            loss += (std::log(sum) - (gold_score - max_score)) / multiThread_miniBatch_data.size();

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    batch_predict_output[inst][i] = batch_predict_output[inst][i] / sum;
                } else {
                    batch_predict_output[inst][i] = 0.0;
                }
            }
            batch_predict_output[inst][gold_act] -= 1.0;
        }

        batch_predict_output /= static_cast<real_t>(multiThread_miniBatch_data.size());

        nnet->Backprop(batch_predict_output);
        nnet->SubsideGradsTo(&gradients, feature_vectors);

        model.update(&batch_cumulated_grads, &adagrad_squares);
        auto end = std::chrono::high_resolution_clock::now();

        if (iter % FLAGS_evaluate_per_iteration == 0) {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000000.0 ;
            std::clog << "[" << iter << "] totally train " << batch_size << " examples, time: " << time_used <<
            " average: " << batch_size / time_used << " examples/second!" << std::endl;
        }

        /*
         * do the evaluation in iteration of training
         * save the best resulting model
         */
        if (iter % FLAGS_evaluate_per_iteration == 0) {
            // do the evaluation
            double dev_uas = test(dev_data, model, nnet.operator*());
            if (dev_uas > best_uas){
                std::ofstream ofs(FLAGS_model_file);
                model.saveModel(ofs);
                ofs.close();

            }
        }
    }


}

//===============================================================================
double DepParser::test(DataSet &test_data, Model<cpu> & model, FeedForwardNNet<gpu> & net) {

    const int num_in = feature_extractor_ptr->getTotalInputSize();
//    const int num_hidden = FLAGS_hidden_size;
    const int num_out = trainsition_system_ptr->action_factory_ptr->total_action_num;

    std::vector<DepParseTree> predict_trees(test_data.size);

    std::vector<DepParseTree> gold_dep_trees;

    for (int inst = 0; inst < test_data.size; ++inst) {

        auto & tree_i = static_cast<DepParseTree& >(*(test_data.outputs[inst]));
        auto & input_i = static_cast<DepParseInput& >(*(test_data.inputs[inst]));

        gold_dep_trees.push_back(tree_i);



        // n shift and n reduce, one more reduce action for root
        int total_act_num_one_sentence = (input_i.size() - 1) * 2;

        /*
         * cache the dependency label in the training set
         */
        std::vector<int> labelIndexCache(tree_i.size);
        int index = 0;
        for (auto iter = tree_i.nodes.begin(); iter != tree_i.nodes.end();
             iter++) {
            int labelIndex = feature_extractor_ptr->getLabelIndex(iter->label);

            if (labelIndex == -1) {
                std::cerr << "Dep label " << iter->label
                << " is not in labelMap!" << std::endl;
                exit(1);
            }

            labelIndexCache[index] = labelIndex;
            index++;
        }

        std::shared_ptr<DepParseState> state_ptr;
        state_ptr.reset(new DepParseState());

        state_ptr->len_ = input_i.size();
        state_ptr->initCache();
//        getCache(input_i);

        //for every state of a sentence
        for (int j = 0; !state_ptr->complete(); j++) {

            TensorContainer<cpu, 2, real_t> input(Shape2(1, num_in));
            TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(1, num_out));

            /*
             * init the input and predict output
             */
            input = 0.0;
            batch_predict_output = 0.0;

            std::vector<int> valid_acts(total_act_num_one_sentence, 0);

            //get current state features
            FeatureVector fv = feature_extractor_ptr->getFeatureVectors(*state_ptr, input_i);
            FeatureVectors fvs;
            fvs.push_back(fv);

            //get current state valid actions
            trainsition_system_ptr->getValidActs(state_ptr.operator*(), valid_acts);

            feature_extractor_ptr->returnInput(fvs, model.featEmbs, input);

            net.Forward(input, batch_predict_output, FLAGS_dropout_prob);


            int opt_act = -1;
            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }
                }
            }

            trainsition_system_ptr->Move(*state_ptr, DepParseShiftReduceActionFactory::action_table[opt_act].operator*());
        }


        // generate the predict tree from the complete state
        trainsition_system_ptr->GenerateOutput( *state_ptr, input_i, predict_trees[inst] );
    }

    DepParseEvalb evalb;
    double result = evalb.evalb(predict_trees, gold_dep_trees);
    return result * 100;
}