#include <sstream>
#include <fstream>
#include <assert.h>
#include <vector>

#include "FeatureEmbedding.h"
#include "Dict.h"

void FeatureEmbedding::init(const real_t init_range){

    static Random<cpu, real_t> rnd(0);  

    rnd.SampleUniform(&data, -1.0 * init_range, init_range);
}

/**
 * read the pre-train embedding for the feature embdding,
 * We only pre-train the word feature embedding
 */
void FeatureEmbedding::readPreTrain(const std::string &file_name, const std::shared_ptr<Dictionary> &dic_ptr){

    const std::tr1::unordered_map<std::string, int> &feature_2_idx = dic_ptr->str_2_index_map;

    std::tr1::unordered_map<std::string, int> pretrain_word_2_idx;
    std::vector<std::vector<real_t>> pretrain_embeddings;
    std::string line;
    std::ifstream in(file_name);

    int index = 0;
    while (getline(in, line)) {
        if (line.empty()) {
            continue;
        }

        std::istringstream iss(line);
        std::vector<real_t> embeddings;

        std::string word;
        real_t d;
        iss >> word;
        while (iss >> d) {
            embeddings.push_back(d);
        }

        pretrain_embeddings.push_back(embeddings);
        pretrain_word_2_idx[word] = index++;
    }

    std::cerr << "### pre-train words size: " << pretrain_embeddings.size() << std::endl;

    // copy the data from cpu to cpu/gpu
    TensorContainer<cpu, 2, real_t> cpu_data(data.shape_);
    Copy(cpu_data, data, data.stream_);

    for (auto& word_index_pair : feature_2_idx) {
        auto pretrain_word_idx_pair = pretrain_word_2_idx.find(word_index_pair.first);

        if (pretrain_word_2_idx.end() != pretrain_word_idx_pair) {  // find it

            int word_index  = word_index_pair.second;
            auto &pre_train_embedding = pretrain_embeddings[pretrain_word_idx_pair->second];

            if (!(word_index >= 0 && word_index < dictionary_size)) {
                std::cerr << "dictSize: " << dictionary_size << std::endl;
                std::cerr << "currWord: " << word_index_pair.first << std::endl;
                std::cerr << "featIndex: " << word_index << std::endl;
            }
            assert (word_index >= 0 && word_index < dictionary_size);
            assert (embedding_dim == static_cast<int>(pre_train_embedding.size()));

            for (int i = 0; i < embedding_dim; i++) {
                cpu_data[word_index][i] = pre_train_embedding[i];
            }
        }
    }

    Copy(data, cpu_data, data.stream_);
}
